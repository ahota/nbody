#include "hip/hip_runtime.h"
#include "nbody.h"

int main(int argc, char **argv) {
    srand(time(NULL));

	printf("Creating bodies...\n");

    float4 *host_pos_mass, *dev_pos_mass;
    float3 *host_acc, *dev_acc;
    float3 *host_output, *dev_output;

	printf("Allocating host memory...\n");

    host_pos_mass = (float4 *)malloc(NUM_BODIES * sizeof(float4));
    host_acc      = (float3 *)malloc(NUM_BODIES * sizeof(float3));
	host_output   = (float3 *)malloc(NUM_BODIES * NUM_STEPS * sizeof(float3));

	printf("Allocating device memory...\n");

	hipMalloc((void **)&dev_pos_mass, NUM_BODIES * sizeof(float4));
    hipMalloc((void **)&dev_acc, NUM_BODIES * sizeof(float3));
    hipMalloc((void **)&dev_output, NUM_BODIES * NUM_STEPS * sizeof(float3));

	printf("Initializing bodies...\n");

	int i;
	for(i = 0; i < NUM_BODIES; i++) {
        host_pos_mass[i].x = rand_coordinate();
        host_pos_mass[i].y = rand_coordinate();
        host_pos_mass[i].z = rand_coordinate();
        host_pos_mass[i].w = rand_mass();
	}

    printf("Initial positions and masses:\n");
    for(i = 0; i < NUM_BODIES; i++) {
        printf("%d:\t%f\t%f\t%f\n", i, host_pos_mass[i].x, host_pos_mass[i].y,
                host_pos_mass[i].z, host_pos_mass[i].w);
    }

	printf("Copying to device...\n");

	hipMemcpy(dev_pos_mass, host_pos_mass, NUM_BODIES * sizeof(float3),
					hipMemcpyHostToDevice);
	hipMemcpy(dev_acc, host_acc, NUM_BODIES * sizeof(float3),
					hipMemcpyHostToDevice);
    hipMemcpy(dev_output, host_output, NUM_BODIES * NUM_STEPS * sizeof(float3),
                    hipMemcpyHostToDevice);

	printf("Running kernel...\n");

    int block_size = (NUM_BODIES < 16) ? 4 : (NUM_BODIES < 256) ? 16 : 32;
    int grid_size  = NUM_BODIES / block_size;
    int mem_size = block_size * sizeof(float3);
    printf("  KERNEL SETTINGS:\n");
    printf("    bodies  = %d\n", NUM_BODIES);
    printf("    tile size = %d\n", block_size);
    printf("    grid size = %d\n", grid_size);
    for(i = 0; i < NUM_STEPS; i++) {
        main_nbody_kernel<<<grid_size, block_size, mem_size>>>(dev_pos_mass,
                dev_acc, dev_output, i);
    }

    printf("Copying to host...\n");

    hipMemcpy(host_output, dev_output, NUM_BODIES * NUM_STEPS * sizeof(float3),
                    hipMemcpyDeviceToHost);
    hipFree(dev_pos_mass);
    hipFree(dev_acc);
    hipFree(dev_output);

    time_t raw_time;
    struct tm *current_time;
    time(&raw_time);
    current_time = localtime(&raw_time);
    char *filename = (char *)malloc(64);
    sprintf(filename, "%02d%02d%02d_%02d%02d%02d.nbd", 
            current_time->tm_year%100, current_time->tm_mon,
            current_time->tm_mday, current_time->tm_hour,
            current_time->tm_min, current_time->tm_sec);

    printf("Saving to %s...\n", filename);

    FILE *outfile = fopen(filename, "w");
    if(outfile == NULL)
        fprintf(stderr, "Error opening file\n");
    else {
        //printf("%f\n", host_output[0].x);
        fprintf(outfile, "%d,%d\n", NUM_BODIES, NUM_STEPS);
        for(i = 0; i < NUM_BODIES * NUM_STEPS; i++) {
            fprintf(outfile, "%f,%f,%f\n", host_output[i].x, host_output[i].y,
                    host_output[i].z);
        }
        fclose(outfile);
    }

	printf("Done.\n");
	return 0;
}

float rand_coordinate() {
    return ((float)rand() / (float)RAND_MAX) * (CMAX - CMIN) + CMIN;
}

float rand_acceleration() {
    return ((float)rand() / (float)RAND_MAX) * (AMAX - AMIN) + AMIN;
}

float rand_mass() { 
    return ((float)rand() / (float)RAND_MAX) * (MMAX - MMIN) + MMIN;
}

__global__ void main_nbody_kernel(float4 *dev_pos_mass, float3 *dev_acc,
        float3 *dev_output, int cur_step) {
    //index into global arrays for this thread's body
    int global_id = blockIdx.x * blockDim.x + threadIdx.x;

    //local copies of this body's position, mass, and acceleration
    float4 my_pos_mass = dev_pos_mass[global_id];
    float3 my_acc = dev_acc[global_id];

    //copy of position and mass for bodies in the current tile
    extern __shared__ float4 tile_pos_mass[]; 

    //iterate over all tiles and update position and acceleration
    //each iteration loads one tile's worth of data from global memory
    //these reads should be coalesced
    int i, tile;
    for(i = 0, tile = 0; i < NUM_BODIES; i += blockDim.x, tile++) {
        //index into global for this thread's body *for this tile*
        int tile_id = tile * blockDim.x + threadIdx.x;

        //threads collaborate to load from global for this tile
        tile_pos_mass[threadIdx.x] = dev_pos_mass[tile_id];
        __syncthreads();

        //update acceleration for this thread's body for this tile
        tile_nbody_kernel(&my_pos_mass, &my_acc);
        __syncthreads();
    }

    //update position for this body
    my_pos_mass.x += my_acc.x;
    my_pos_mass.y += my_acc.y;
    my_pos_mass.z += my_acc.z;

    //update global position array
    dev_pos_mass[global_id] = my_pos_mass;

    //update global output
    dev_output[cur_step * NUM_BODIES + global_id].x = my_pos_mass.x;
    dev_output[cur_step * NUM_BODIES + global_id].y = my_pos_mass.y;
    dev_output[cur_step * NUM_BODIES + global_id].z = my_pos_mass.z;
}

__device__ void tile_nbody_kernel(float4 *my_pos_mass, float3 *my_acc) {
    //tile position array from the outer kernel
    //pre-loaded with this tile's positions and masses
    extern __shared__ float4 tile_pos_mass[];

    //iterate over each body in the tile and calculate its effect on
    //this thread's body
    int i;
    for(i = 0; i < blockDim.x; i++) {
        force_kernel(my_pos_mass, &tile_pos_mass[i], my_acc);
    }
}

__device__ void force_kernel(float4 *body_i, float4 *body_j, float3 *acc_i) {
    //calculate distance components
    float3 d;
    d.x = body_i->x - body_j->x;
    d.y = body_i->y - body_j->y;
    d.z = body_i->z - body_j->z;

    //use episilon softener
    //  r^2 + epsilon^2
    float denominator = d.x * d.x + d.y * d.y + d.z * d.z + EPSILON2;
    //cube and sqrt to get (r^2 + epsilon^2)^(3/2)
    denominator = sqrt( denominator * denominator * denominator );

    float acc = body_j->w / denominator;

    //update acceleration
    acc_i->x += acc * d.x;
    acc_i->y += acc * d.y;
    acc_i->z += acc * d.z;
}
